#include "hip/hip_runtime.h"
#include "CudaRender.cuh"

struct cMaterial
{
    enum Type { Lambert, Metal, Dielectric };
    Type type;
    float3 albedo;
    float3 emissive;
    float roughness;
    float ri;
};

__device__ float sqLength(const float3& v)
{
    return v.x*v.x + v.y*v.y + v.z*v.z;
}

inline __device__ void AssertUnit(const float3& v)
{
    assert(fabsf(sqLength(v) - 1.0f) < 0.01f);
}

__device__ bool HitSphere(const cRay& r, const cSphere& s, float tMin, float tMax, float& outHitT)
{
    AssertUnit(r.dir);
    float3 oc = r.orig - s.center;
    float b = dot(oc, r.dir);
    float c = dot(oc, oc) - s.radius*s.radius;
    float discr = b * b - c;
    if (discr > 0)
    {
        float discrSq = sqrtf(discr);

        float t = (-b - discrSq);
        if (t < tMax && t > tMin)
        {
            outHitT = t;
            return true;
        }
        t = (-b + discrSq);
        if (t < tMax && t > tMin)
        {
            outHitT = t;
            return true;
        }
    }
    return false;
}

__global__ void HitWorldKernel(const DeviceData data, float tMin, float tMax)
{
    const int rIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if (rIdx >= data.numRays)
        return;

    const cRay& r = data.rays[rIdx];
    if (r.isDone())
        return;

    int hitId = -1;
    float closest = tMax, hitT;
    for (int i = 0; i < data.spheresCount; ++i)
    {
        if (HitSphere(r, data.spheres[i], tMin, closest, hitT))
        {
            closest = hitT;
            hitId = i;
        }
    }

    data.hits[rIdx] = cHit(closest, hitId);
}

void initDeviceData(const Sphere* spheres, const int spheresCount, const int numRays, DeviceData& data)
{
    data.numRays = numRays;
    data.spheresCount = spheresCount;

    // allocate device memory
    hipMalloc((void**)&data.spheres, spheresCount * sizeof(cSphere));
    hipMalloc((void**)&data.rays, numRays * sizeof(cRay));
    hipMalloc((void**)&data.hits, numRays * sizeof(cHit));

    // copy spheres to device
    hipMemcpy(data.spheres, spheres, spheresCount * sizeof(cSphere), hipMemcpyHostToDevice);
}

void HitWorldDevice(const Ray* rays, float tMin, float tMax, Hit* hits, DeviceData data)
{
    // copy rays to device
    hipMemcpy(data.rays, rays, data.numRays * sizeof(cRay), hipMemcpyHostToDevice);

    // call kernel
    const int threadsPerBlock = 1024;
    const int blocksPerGrid = ceilf((float)data.numRays / threadsPerBlock);

    HitWorldKernel <<<blocksPerGrid, threadsPerBlock >>> (data, tMin, tMax);

    // copy hits to host
    hipMemcpy(hits, data.hits, data.numRays * sizeof(cHit), hipMemcpyDeviceToHost);
}


void freeDeviceData(const DeviceData& data)
{
    hipFree(data.spheres);
    hipFree(data.rays);
    hipFree(data.hits);
}

__device__ uint cXorShift32(uint& state)
{
    uint x = state;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 15;
    state = x;
    return x;
}

__device__ float cRandomFloat01(uint& state)
{
    return (cXorShift32(state) & 0xFFFFFF) / 16777216.0f;
}

__device__ float3 cRandomUnitVector(uint& state)
{
    float z = cRandomFloat01(state) * 2.0f - 1.0f;
    float a = cRandomFloat01(state) * 2.0f * kPI;
    float r = sqrtf(1.0f - z * z);
    float x = r * cosf(a);
    float y = r * sinf(a);
    return make_float3(x, y, z);
}

__device__ float3 cRandomInUnitSphere(uint& state)
{
    float3 p;
    do {
        p = 2.0*make_float3(cRandomFloat01(state) - 1, cRandomFloat01(state) - 1, cRandomFloat01(state) - 1);
    } while (sqLength(p) >= 1.0);
    return p;
}

__device__ bool refract(const float3& v, const float3& n, float nint, float3& outRefracted)
{
    AssertUnit(v);
    float dt = dot(v, n);
    float discr = 1.0f - nint * nint*(1 - dt * dt);
    if (discr > 0)
    {
        outRefracted = nint * (v - n * dt) - n * sqrtf(discr);
        return true;
    }
    return false;
}

__device__ float cSchlick(float cosine, float ri)
{
    float r0 = (1 - ri) / (1 + ri);
    r0 = r0 * r0;
    return r0 + (1 - r0)*powf(1 - cosine, 5);
}

__device__ bool ScatterNoLightSampling(const DeviceData& data, const cMaterial& mat, const cRay& r_in, const cHit& rec, float3& attenuation, cRay& scattered, uint& state)
{
    const float3 hitPos = r_in.pointAt(rec.t);
    const float3 hitNormal = data.spheres[rec.id].normalAt(hitPos);

    if (mat.type == cMaterial::Lambert)
    {
        // random point on unit sphere that is tangent to the hit point
        float3 target = hitPos + hitNormal + cRandomUnitVector(state);
        scattered = cRay(hitPos, normalize(target - hitPos));
        attenuation = mat.albedo;

        return true;
    }
    else if (mat.type == cMaterial::Metal)
    {
        AssertUnit(r_in.dir); AssertUnit(hitNormal);
        float3 refl = reflect(r_in.dir, hitNormal);
        // reflected ray, and random inside of sphere based on roughness
        float roughness = mat.roughness;
        scattered = cRay(hitPos, normalize(refl + roughness * cRandomInUnitSphere(state)));
        attenuation = mat.albedo;
        return dot(scattered.dir, hitNormal) > 0;
    }
    else if (mat.type == cMaterial::Dielectric)
    {
        AssertUnit(r_in.dir); AssertUnit(hitNormal);
        float3 outwardN;
        float3 rdir = r_in.dir;
        float3 refl = reflect(rdir, hitNormal);
        float nint;
        attenuation = make_float3(1, 1, 1);
        float3 refr;
        float reflProb;
        float cosine;
        if (dot(rdir, hitNormal) > 0)
        {
            outwardN = -1*hitNormal;
            nint = mat.ri;
            cosine = mat.ri * dot(rdir, hitNormal);
        }
        else
        {
            outwardN = hitNormal;
            nint = 1.0f / mat.ri;
            cosine = -dot(rdir, hitNormal);
        }
        if (refract(rdir, outwardN, nint, refr))
        {
            reflProb = cSchlick(cosine, mat.ri);
        }
        else
        {
            reflProb = 1;
        }
        if (cRandomFloat01(state) < reflProb)
            scattered = cRay(hitPos, normalize(refl));
        else
            scattered = cRay(hitPos, normalize(refr));
    }
    else
    {
        attenuation = make_float3(1, 0, 1);
        return false;
    }
    return true;
}
