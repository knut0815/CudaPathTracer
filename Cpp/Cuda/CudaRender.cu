#include "hip/hip_runtime.h"
#include "CudaRender.cuh"
#include "../Source/Config.h"

__device__ float sqLength(const float3& v)
{
    return v.x*v.x + v.y*v.y + v.z*v.z;
}

inline __device__ void AssertUnit(const float3& v)
{
    assert(fabsf(sqLength(v) - 1.0f) < 0.01f);
}

__device__ bool HitSphere(const cRay& r, const cSphere& s, float tMin, float tMax, float& outHitT)
{
    AssertUnit(r.dir);
    float3 oc = r.orig - s.center;
    float b = dot(oc, r.dir);
    float c = dot(oc, oc) - s.radius*s.radius;
    float discr = b * b - c;
    if (discr > 0)
    {
        float discrSq = sqrtf(discr);

        float t = (-b - discrSq);
        if (t < tMax && t > tMin)
        {
            outHitT = t;
            return true;
        }
        t = (-b + discrSq);
        if (t < tMax && t > tMin)
        {
            outHitT = t;
            return true;
        }
    }
    return false;
}

__global__ void HitWorldKernel(const DeviceData data, float tMin, float tMax)
{
    const int rIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if (rIdx >= data.numRays)
        return;

    const cRay& r = data.rays[rIdx];
    if (r.isDone())
        return;

    int hitId = -1;
    float closest = tMax, hitT;
    for (int i = 0; i < data.spheresCount; ++i)
    {
        if (HitSphere(r, data.spheres[i], tMin, closest, hitT))
        {
            closest = hitT;
            hitId = i;
        }
    }

    data.hits[rIdx] = cHit(closest, hitId);
}

__device__ uint cXorShift32(uint& state)
{
    uint x = state;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 15;
    state = x;
    return x;
}

__device__ float cRandomFloat01(uint& state)
{
    return (cXorShift32(state) & 0xFFFFFF) / 16777216.0f;
}

__device__ float3 cRandomUnitVector(uint& state)
{
    float z = cRandomFloat01(state) * 2.0f - 1.0f;
    float a = cRandomFloat01(state) * 2.0f * kPI;
    float r = sqrtf(1.0f - z * z);
    float x = r * cosf(a);
    float y = r * sinf(a);
    return make_float3(x, y, z);
}

__device__ float3 cRandomInUnitSphere(uint& state)
{
    float3 p;
    do {
        p = make_float3(2*cRandomFloat01(state) - 1, 2*cRandomFloat01(state) - 1, 2*cRandomFloat01(state) - 1);
    } while (sqLength(p) >= 1.0);
    return p;
}

/*
* based off http://www.reedbeta.com/blog/quick-and-easy-gpu-random-numbers-in-d3d11/
*/
__device__ uint cWang_hash(uint seed)
{
    seed = (seed ^ 61) ^ (seed >> 16);
    seed *= 9;
    seed = seed ^ (seed >> 4);
    seed *= 0x27d4eb2d;
    seed = seed ^ (seed >> 15);
    return seed;
}

__device__ bool refract(const float3& v, const float3& n, float nint, float3& outRefracted)
{
    AssertUnit(v);
    float dt = dot(v, n);
    float discr = 1.0f - nint * nint*(1 - dt * dt);
    if (discr > 0)
    {
        outRefracted = nint * (v - n * dt) - n * sqrtf(discr);
        return true;
    }
    return false;
}

__device__ float cSchlick(float cosine, float ri)
{
    float r0 = (1 - ri) / (1 + ri);
    r0 = r0 * r0;
    return r0 + (1 - r0)*powf(1 - cosine, 5);
}

__device__ bool ScatterNoLightSampling(const DeviceData& data, const cMaterial& mat, const cRay& r_in, const cHit& rec, float3& attenuation, cRay& scattered, uint& state)
{
    const float3 hitPos = r_in.pointAt(rec.t);
    const float3 hitNormal = data.spheres[rec.id].normalAt(hitPos);

    if (mat.type == cMaterial::Lambert)
    {
        // random point on unit sphere that is tangent to the hit point
        float3 target = hitPos + hitNormal + cRandomUnitVector(state);
        scattered = cRay(hitPos, normalize(target - hitPos));
        attenuation = mat.albedo;

        return true;
    }
    else if (mat.type == cMaterial::Metal)
    {
        AssertUnit(r_in.dir); AssertUnit(hitNormal);
        float3 refl = reflect(r_in.dir, hitNormal);
        // reflected ray, and random inside of sphere based on roughness
        float roughness = mat.roughness;
        scattered = cRay(hitPos, normalize(refl + roughness * cRandomInUnitSphere(state)));
        attenuation = mat.albedo;
        return dot(scattered.dir, hitNormal) > 0;
    }
    else if (mat.type == cMaterial::Dielectric)
    {
        AssertUnit(r_in.dir); AssertUnit(hitNormal);
        float3 outwardN;
        float3 rdir = r_in.dir;
        float3 refl = reflect(rdir, hitNormal);
        float nint;
        attenuation = make_float3(1, 1, 1);
        float3 refr;
        float reflProb;
        float cosine;
        if (dot(rdir, hitNormal) > 0)
        {
            outwardN = -1*hitNormal;
            nint = mat.ri;
            cosine = mat.ri * dot(rdir, hitNormal);
        }
        else
        {
            outwardN = hitNormal;
            nint = 1.0f / mat.ri;
            cosine = -dot(rdir, hitNormal);
        }
        if (refract(rdir, outwardN, nint, refr))
        {
            reflProb = cSchlick(cosine, mat.ri);
        }
        else
        {
            reflProb = 1;
        }
        if (cRandomFloat01(state) < reflProb)
            scattered = cRay(hitPos, normalize(refl));
        else
            scattered = cRay(hitPos, normalize(refr));
    }
    else
    {
        attenuation = make_float3(1, 0, 1);
        return false;
    }
    return true;
}

__global__ void ScatterKernel(const DeviceData data, const uint depth)
{
    const int rIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if (rIdx >= data.numRays)
        return;

    const cRay& r = data.rays[rIdx];
    if (r.isDone())
        return;

    uint state = (cWang_hash(rIdx) + (data.frame*kMaxDepth + depth) * 101141101) * 336343633;

    const cHit& hit = data.hits[rIdx];
    cSample& sample = data.samples[rIdx];
    if (depth == 0)
    {
        sample.color = make_float3(0);
        sample.attenuation = make_float3(1);
    }

    if (hit.id >= 0)
    {
        cRay scattered;
        const cMaterial& mat = data.materials[hit.id];
        float3 local_attenuation;
        sample.color += mat.emissive * sample.attenuation;
        if (depth < kMaxDepth && ScatterNoLightSampling(data, mat, r, hit, local_attenuation, scattered, state))
        {
            sample.attenuation *= local_attenuation;
            data.rays[rIdx] = scattered;
        }
        else
        {
            data.rays[rIdx].setDone();
        }
    }
    else
    {
        // sky
        float3 unitDir = r.dir;
        float t = 0.5f*(unitDir.y + 1.0f);
        sample.color += sample.attenuation * ((1.0f - t)*make_float3(1) + t * make_float3(0.5f, 0.7f, 1.0f)) * 0.3f;
        data.rays[rIdx].setDone();
    }
}

void deviceInitData(const Sphere* spheres, const Material* materials, const int spheresCount, const int numRays, DeviceData& data)
{
    data.numRays = numRays;
    data.spheresCount = spheresCount;

    // allocate device memory
    hipMalloc((void**)&data.spheres, spheresCount * sizeof(cSphere));
    hipMalloc((void**)&data.materials, spheresCount * sizeof(cMaterial));
    hipMalloc((void**)&data.rays, numRays * sizeof(cRay));
    hipMalloc((void**)&data.hits, numRays * sizeof(cHit));
    hipMalloc((void**)&data.samples, numRays * sizeof(cSample));

    // copy spheres and materials to device
    hipMemcpy(data.spheres, spheres, spheresCount * sizeof(cSphere), hipMemcpyHostToDevice);
    hipMemcpy(data.materials, materials, spheresCount * sizeof(cMaterial), hipMemcpyHostToDevice);
}

void deviceStartFrame(const Ray* rays, const uint frame, DeviceData& data) {
    data.frame = frame;
    // copy rays to device
    hipMemcpy(data.rays, rays, data.numRays * sizeof(cRay), hipMemcpyHostToDevice);
}

void deviceRenderFrame(const float tMin, const float tMax, const uint depth, const DeviceData data)
{
    // call kernel
    const int threadsPerBlock = 1024;
    const int blocksPerGrid = ceilf((float)data.numRays / threadsPerBlock);

    HitWorldKernel <<<blocksPerGrid, threadsPerBlock >> > (data, tMin, tMax);
    ScatterKernel <<<blocksPerGrid, threadsPerBlock >> > (data, depth);
}

void deviceEndFrame(Sample* samples, const DeviceData& data)
{
    // copy samples to host
    hipMemcpy(samples, data.samples, data.numRays * sizeof(cSample), hipMemcpyDeviceToHost);
}

void deviceFreeData(const DeviceData& data)
{
    hipFree(data.spheres);
    hipFree(data.rays);
    hipFree(data.hits);
    hipFree(data.samples);
}
